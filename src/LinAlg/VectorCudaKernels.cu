#include "hip/hip_runtime.h"
// Copyright (c) 2017, Lawrence Livermore National Security, LLC.
// Produced at the Lawrence Livermore National Laboratory (LLNL).
// LLNL-CODE-742473. All rights reserved.
//
// This file is part of HiOp. For details, see https://github.com/LLNL/hiop. HiOp 
// is released under the BSD 3-clause license (https://opensource.org/licenses/BSD-3-Clause). 
// Please also read "Additional BSD Notice" below.
//
// Redistribution and use in source and binary forms, with or without modification, 
// are permitted provided that the following conditions are met:
// i. Redistributions of source code must retain the above copyright notice, this list 
// of conditions and the disclaimer below.
// ii. Redistributions in binary form must reproduce the above copyright notice, 
// this list of conditions and the disclaimer (as noted below) in the documentation and/or 
// other materials provided with the distribution.
// iii. Neither the name of the LLNS/LLNL nor the names of its contributors may be used to 
// endorse or promote products derived from this software without specific prior written 
// permission.
//
// THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND ANY 
// EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED WARRANTIES 
// OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT 
// SHALL LAWRENCE LIVERMORE NATIONAL SECURITY, LLC, THE U.S. DEPARTMENT OF ENERGY OR 
// CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR 
// CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS 
// OR SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED 
// AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT 
// (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE, 
// EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
//
// Additional BSD Notice
// 1. This notice is required to be provided under our contract with the U.S. Department 
// of Energy (DOE). This work was produced at Lawrence Livermore National Laboratory under 
// Contract No. DE-AC52-07NA27344 with the DOE.
// 2. Neither the United States Government nor Lawrence Livermore National Security, LLC 
// nor any of their employees, makes any warranty, express or implied, or assumes any 
// liability or responsibility for the accuracy, completeness, or usefulness of any 
// information, apparatus, product, or process disclosed, or represents that its use would
// not infringe privately-owned rights.
// 3. Also, reference herein to any specific commercial products, process, or services by 
// trade name, trademark, manufacturer or otherwise does not necessarily constitute or 
// imply its endorsement, recommendation, or favoring by the United States Government or 
// Lawrence Livermore National Security, LLC. The views and opinions of authors expressed 
// herein do not necessarily state or reflect those of the United States Government or 
// Lawrence Livermore National Security, LLC, and shall not be used for advertising or 
// product endorsement purposes.

/**
 * @file VectorCudaKernels.cu
 *
 * @author Nai-Yuan Chiang <chiang7@llnl.gov>, LLNL
 *
 */
#include "VectorCudaKernels.hpp"
#include <hip/hip_runtime.h>
#include <hipblas.h>

#include <thrust/device_ptr.h>
#include <thrust/device_vector.h>
#include <thrust/transform.h>
#include <thrust/functional.h>
#include <thrust/fill.h>
#include <thrust/replace.h>
#include <thrust/transform_reduce.h>
#include <thrust/extrema.h>
#include <thrust/logical.h>
#include <thrust/execution_policy.h>
#include <thrust/device_malloc.h>
#include <thrust/device_free.h>


//#include <cmath>
//#include <limits>

/// @brief compute abs(b-a)
template <typename T>
struct thrust_abs_diff: public thrust::binary_function<T,T,T>
{
    __host__ __device__
    T operator()(const T& a, const T& b)
    {
        return fabs(b - a);
    }
};

/// @brief compute abs(a)
template <typename T>
struct thrust_abs: public thrust::unary_function<T,T>
{
    __host__ __device__
    T operator()(const T& a)
    {
        return fabs(a);
    }
};

/// @brief return true if abs(a) < tol_
struct thrust_abs_less
{
    const double tol_;
    thrust_abs_less(double tol) : tol_(tol) {}
    
    __host__ __device__
    int operator()(const double& a)
    {
        return (fabs(a) < tol_);
    }
};

/// @brief return true if a < tol_
struct thrust_less
{
    const double tol_;
    thrust_less(double tol) : tol_(tol) {}
    
    __host__ __device__
    int operator()(const double& a)
    {
        return (a < tol_);
    }
};

/// @brief return true if (0.0 < a) - (a < 0.0)
template <typename T>
struct thrust_sig: public thrust::unary_function<T,T>
{
    __host__ __device__
    T operator()(const T& a)
    {
        return static_cast<double>( (0.0 < a) - (a < 0.0) ); 
    }
};

/// @brief compute sqrt(a)
template <typename T>
struct thrust_sqrt: public thrust::unary_function<T,T>
{
    __host__ __device__
    T operator()(const T& a)
    {
        return sqrt(a); 
    }
};

/// @brief compute log(a) if a > 0, otherwise returns 0
template <typename T>
struct thrust_log_select: public thrust::unary_function<T,double>
{
    __host__ __device__
    double operator()(const T& a)
    {
        if(a>0){
          return log(a);
        }
        return 0.; 
    }
};

/// @brief compute isinf(a)
template <typename T>
struct thrust_isinf: public thrust::unary_function<T,bool>
{
    __host__ __device__
    bool operator()(const T& a)
    {
      return isinf(a);
    }
};

/// @brief compute isfinite(a)
template <typename T>
struct thrust_isfinite: public thrust::unary_function<T,bool>
{
    __host__ __device__
    bool operator()(const T& a)
    {
      return isfinite(a);
    }
};

/// @brief compute a==0.0
template <typename T>
struct thrust_iszero: public thrust::unary_function<T,bool>
{
    __host__ __device__
    bool operator()(const T& a)
    {
      return a== (T) (0.0);
    }
};

/// @brief compute isnan(a)
template <typename T>
struct thrust_isnan: public thrust::unary_function<T,bool>
{
    __host__ __device__
    bool operator()(const T& a)
    {
      return isnan(a);
    }
};

/// @brief compute (bool) (a)
struct thrust_istrue : public thrust::unary_function<int, bool>
{
    __host__ __device__
    bool operator()(const int& a)
    {
      return a;
    }
};

/** @brief Set y[i] = min(y[i],c), for i=[0,n_local-1] */
__global__ void component_min_cu(int n, double* vec, const double c)
{
  const int num_threads = blockDim.x * gridDim.x;
  const int tid = blockIdx.x * blockDim.x + threadIdx.x;    
  for (int i = tid; i < n; i += num_threads) {
    y[i] = (y[i]<c) ? y[i] : c;	
  }
}

/** @brief Set y[i] = min(y[i],x[i]), for i=[0,n_local-1] */
__global__ void component_min_cu(int n, double* y, const double* x)
{
  const int num_threads = blockDim.x * gridDim.x;
  const int tid = blockIdx.x * blockDim.x + threadIdx.x;    
  for (int i = tid; i < n; i += num_threads) {
    y[i] = (y[i]<x[i]) ? y[i] : x[i];	
  }
}

/** @brief Set y[i] = max(y[i],c), for i=[0,n_local-1] */
__global__ void component_max_cu(int n, double* y, const double c)
{
  const int num_threads = blockDim.x * gridDim.x;
  const int tid = blockIdx.x * blockDim.x + threadIdx.x;    
  for (int i = tid; i < n; i += num_threads) {
    y[i] = (y[i]>c) ? y[i] : c;	
  }
}

/** @brief Set y[i] = max(y[i],x[i]), for i=[0,n_local-1] */
__global__ void component_max_cu(int n, double* y, const double* x)
{
  const int num_threads = blockDim.x * gridDim.x;
  const int tid = blockIdx.x * blockDim.x + threadIdx.x;    
  for (int i = tid; i < n; i += num_threads) {
    y[i] = (y[i]>x[i]) ? y[i] : x[i];	
  }
}

/// @brief Copy from src the elements specified by the indices in id. 
__global__ void copy_from_index_cu(int n, double* vec, const double* val, const int* id)
{
  const int num_threads = blockDim.x * gridDim.x;
  const int tid = blockIdx.x * blockDim.x + threadIdx.x;    
  for (int i = tid; i < n; i += num_threads) {
    vec[i] = val[id[i]];	
  }
}

/// @brief Performs axpy, y += alpha*x, on the indexes in this specified by id.
__global__ void axpy_w_map_cu(int n, double* yd, const double* xd, const int* id, double alpha)
{
  const int num_threads = blockDim.x * gridDim.x;
  const int tid = blockIdx.x * blockDim.x + threadIdx.x;    
  for (int i = tid; i < n; i += num_threads) {
    assert(id[i]<n);
    yd[id[i]] = alpha * xd[i] + yd[id[i]];
  }
}

/** @brief this[i] += alpha*x[i]*z[i] forall i */
__global__ void axzpy_cu(int n, double* yd, const double* xd, const double* zd, double alpha)
{

  const int num_threads = blockDim.x * gridDim.x;
  const int tid = blockIdx.x * blockDim.x + threadIdx.x;    
  for (int i = tid; i < n; i += num_threads) {
    yd[i] = alpha * xd[i] * zd[i] + yd[i];
  }
}

/** @brief this[i] += alpha*x[i]/z[i] forall i */
__global__ void axdzpy_cu(int n, double* yd, const double* xd, const double* zd, double alpha)
{

  const int num_threads = blockDim.x * gridDim.x;
  const int tid = blockIdx.x * blockDim.x + threadIdx.x;    
  for (int i = tid; i < n; i += num_threads) {
    yd[i] = alpha * xd[i] / zd[i] + yd[i];
  }
}

/** @brief this[i] += alpha*x[i]/z[i] forall i with pattern selection */
__global__ void axdzpy_w_pattern_cu(int n, double* yd, const double* xd, const double* zd, const double* id, double alpha)
{

  const int num_threads = blockDim.x * gridDim.x;
  const int tid = blockIdx.x * blockDim.x + threadIdx.x;    
  for (int i = tid; i < n; i += num_threads) {
    if(id[i] == 1.0) {
      yd[i] = alpha * xd[i] / zd[i] + yd[i];
    }
  }
}

/** @brief y[i] += alpha*1/x[i] + y[i] forall i with pattern selection */
__global__ void adxpy_w_pattern_cu(int n, double* yd, const double* xd, const double* id, double alpha)
{

  const int num_threads = blockDim.x * gridDim.x;
  const int tid = blockIdx.x * blockDim.x + threadIdx.x;    
  for (int i = tid; i < n; i += num_threads) {
    if(id[i]==1.0) {
      yd[i] = alpha / xd[i] + yd[i];
    }
  }
}

/**  @brief  elements of this that corespond to nonzeros in ix are divided by elements of v.
     The rest of elements of this are set to zero.*/
__global__ void component_div_w_pattern_cu(int n, double* yd, const double* xd, const double* id)
{

  const int num_threads = blockDim.x * gridDim.x;
  const int tid = blockIdx.x * blockDim.x + threadIdx.x;    
  for (int i = tid; i < n; i += num_threads) {
    if(id[i]==1.0) {
      yd[i] = yd[i] / xd[i];
    } else {
      yd[i] = 0.0;
    }
  }
}

/** @brief y[i] += c forall i */
__global__ void add_constant_cu(int n, double* yd, double c)
{
  const int num_threads = blockDim.x * gridDim.x;
  const int tid = blockIdx.x * blockDim.x + threadIdx.x;    
  for (int i = tid; i < n; i += num_threads) {
    yd[i] =  yd[i] + c;
  }
}

/** @brief y[i] += c forall i with pattern selection */
__global__ void add_constant_w_pattern_cu(int n, double* yd, double c, const double* id)
{
  const int num_threads = blockDim.x * gridDim.x;
  const int tid = blockIdx.x * blockDim.x + threadIdx.x;    
  for (int i = tid; i < n; i += num_threads) {
    yd[i] =  yd[i] + c * id[i];
  }
}

/// @brief Invert (1/x) the elements of this
__global__ void invert_cu(int n, double* yd)
{
  const int num_threads = blockDim.x * gridDim.x;
  const int tid = blockIdx.x * blockDim.x + threadIdx.x;    
  for (int i = tid; i < n; i += num_threads) {
    yd[i] =  1. / yd[i];
  }
}

/** @brief Linear damping term */
__global__ void set_linear_damping_term_cu(int n, double* yd, const double* vd, const double* ld, const double* rd)
{
  const int num_threads = blockDim.x * gridDim.x;
  const int tid = blockIdx.x * blockDim.x + threadIdx.x;    
  for (int i = tid; i < n; i += num_threads) {
    if(ld[i]==1.0 && rd[i]==0.0) {
      yd[i] = vd[i];
    } else {
      yd[i] = 0.0;
    }
  }
}

/** 
* @brief Performs `this[i] = alpha*this[i] + sign*ct` where sign=1 when EXACTLY one of 
* ixleft[i] and ixright[i] is 1.0 and sign=0 otherwise. 
*/
__global__ void add_linear_damping_term_cu(int n, double* data, const double* ixl, const double* ixr, double alpha, double ct)
{
  const int num_threads = blockDim.x * gridDim.x;
  const int tid = blockIdx.x * blockDim.x + threadIdx.x;    
  for (int i = tid; i < n; i += num_threads) {
    data[i] = alpha * data[i] + ct*(ixl[i]-ixr[i]);
  }
}

/** @brief y[i] = 1.0 if x[i] is positive and id[i] = 1.0, otherwise y[i] = 0 */
__global__ void is_posive_w_pattern_cu(int n, double* data, const double* vd, const double* id)
{
  const int num_threads = blockDim.x * gridDim.x;
  const int tid = blockIdx.x * blockDim.x + threadIdx.x;    
  for (int i = tid; i < n; i += num_threads) {
    data[i] = (id[i] == 1.0 && vd[i] > 0.0) ? 1 : 0;
  }
}

/** @brief y[i] = x[i] if id[i] = 1.0, otherwise y[i] = val_else */
__global__ void set_val_w_pattern_cu(int n, double* data, const double* vd, const double* id, double val_else)
{
  const int num_threads = blockDim.x * gridDim.x;
  const int tid = blockIdx.x * blockDim.x + threadIdx.x;    
  for (int i = tid; i < n; i += num_threads) {
    data[i] = (id[i] == 1.0) ? vd[i] : val_else;
  }
}

/** @brief data[i] = 0 if id[i]==0.0 */
__global__ void select_pattern_cu(int n, double* data, const double* id)
{
  const int num_threads = blockDim.x * gridDim.x;
  const int tid = blockIdx.x * blockDim.x + threadIdx.x;    
  for (int i = tid; i < n; i += num_threads) {
    if(id[i] == 0.0) {
      data[i] = 0.0;
    }    
  }
}

__global__ void match_pattern_cu(int n, double* data, const double* id)
{
  const int num_threads = blockDim.x * gridDim.x;
  const int tid = blockIdx.x * blockDim.x + threadIdx.x;    
  for (int i = tid; i < n; i += num_threads) {
    if(id[i] == 0.0) {
      data[i] = 0.0;
    }    
  }
}

/** @brief Project solution into bounds  */
__global__ void project_into_bounds_cu(int n,
                                       double* xd,
                                       const double* xld,
                                       const double* ild,
                                       const double* xud,
                                       const double* iud,
                                       double kappa1,
                                       double kappa2,
                                       double small_real)
{
  const int num_threads = blockDim.x * gridDim.x;
  const int tid = blockIdx.x * blockDim.x + threadIdx.x;

  for (int i = tid; i < n; i += num_threads) {
    double aux  = 0.0;
    double aux2 = 0.0;
    if(ild[i] != 0.0 && iud[i] != 0.0) {
      aux = kappa2*(xud[i] - xld[i]) - small_real;
      aux2 = xld[i] + fmin(kappa1 * fmax(1.0, fabs(xld[i])), aux);
      if(xd[i] < aux2) {
        xd[i] = aux2;
      } else {
        aux2 = xud[i] - fmin(kappa1 * fmax(1.0, fabs(xud[i])), aux);
        if(xd[i] > aux2) {
          xd[i] = aux2;
        }
      }
#ifdef HIOP_DEEPCHECKS
      assert(xd[i] > xld[i] && xd[i] < xud[i] && "this should not happen -> HiOp bug");
#endif
    } else {
      if(ild[i] != 0.0) {
        xd[i] = fmax(xd[i], xld[i] + kappa1*fmax(1.0, fabs(xld[i])) - small_real); 
      }
      if(iud[i] != 0.0) {
        xd[i] = fmin(xd[i], xud[i] - kappa1*fmax(1.0, fabs(xud[i])) - small_real);
      } else { 
        /*nothing for free vars  */
      }
    }
  }
}

/** @brief max{a\in(0,1]| x+ad >=(1-tau)x} */
__global__ void fraction_to_the_boundry_cu(int n, double* yd, const double* xd, const double* dd, double tau)
{
  const int num_threads = blockDim.x * gridDim.x;
  const int tid = blockIdx.x * blockDim.x + threadIdx.x;    
  for (int i = tid; i < n; i += num_threads) {
    if(dd[i]>=0) {
      yd[i] = 1.0;
    } else {
      yd[i] = -tau*xd[i]/dd[i];
    }
  }
}

/** @brief max{a\in(0,1]| x+ad >=(1-tau)x} with pattern select */
__global__ void fraction_to_the_boundry_w_pattern_cu(int n,
                                                     double* yd,
                                                     const double* xd,
                                                     const double* dd,
                                                     const double* id,
                                                     double tau)
{
  const int num_threads = blockDim.x * gridDim.x;
  const int tid = blockIdx.x * blockDim.x + threadIdx.x;    
  for (int i = tid; i < n; i += num_threads) {
    if(dd[i]>=0 || id[i]==0) {
      yd[i] = 1.0;
    } else {
      yd[i] = -tau*xd[i]/dd[i];
    }
  }
}

/** @brief y[i] = 0 if id[i]==0.0 && xd[i]!=0.0, otherwise y[i] = 1*/
__global__ void set_match_pattern_cu(int n, int* yd, const double* xd, const double* id)
{
  const int num_threads = blockDim.x * gridDim.x;
  const int tid = blockIdx.x * blockDim.x + threadIdx.x;    
  for (int i = tid; i < n; i += num_threads) {
    if(id[i]==0.0 && xd[i]!=0.0) {
      yd[i] = 0;
    } else {
      yd[i] = 1;
    }
  }
}

/** @brief Adjusts duals. */
__global__ void adjust_duals_cu(int n, double* zd, const double* xd, const double* id, double mu, double kappa)
{
  const int num_threads = blockDim.x * gridDim.x;
  const int tid = blockIdx.x * blockDim.x + threadIdx.x;    
  double a,b;
  for (int i = tid; i < n; i += num_threads) {
    // preemptive loop to reduce number of iterations?
    if(id[i] == 1.) {
      // precompute a and b in another loop?
      a = mu/xd[i];
      b = a/kappa;
      a = a*kappa;
      // Necessary conditionals
      if(zd[i]<b) {
        zd[i] = b;
      } else {
        //zd[i]>=b
        if(a<=b) { 
          zd[i] = b;
        } else {
          //a>b
          if(a<zd[i]) {
            zd[i] = a;
          }
        }
      }
      // - - - - 
      //else a>=z[i] then *z=*z (z[i] does not need adjustment)
    }
  }
}

/// set nonlinear type
__global__ void set_nonlinear_type_cu(const int n,
                                      const int length,
                                      hiop::hiopInterfaceBase::NonlinearityType* arr,
                                      const int start,
                                      const hiop::hiopInterfaceBase::NonlinearityType* arr_src,
                                      const int start_src)
{
  const int num_threads = blockDim.x * gridDim.x;
  const int tid = blockIdx.x * blockDim.x + threadIdx.x;    
  for (int i = tid; i < n && i < length; i += num_threads) {
    arr[start+i] = arr_src[start_src+i];
  }
}

/// set nonlinear type
__global__ void set_nonlinear_type_cu(const int n,
                                      const int length,
                                      hiop::hiopInterfaceBase::NonlinearityType* arr,
                                      const int start,
                                      const hiop::hiopInterfaceBase::NonlinearityType arr_src)
{
  const int num_threads = blockDim.x * gridDim.x;
  const int tid = blockIdx.x * blockDim.x + threadIdx.x;    
  for (int i = tid; i < n && i < length; i += num_threads) {
    arr[start+i] = arr_src;
  }
}

/// for hiopVectorIntCuda
/**
 * @brief Set the vector entries to be a linear space of starting at i0 containing evenly 
 * incremented integers up to i0+(n-1)di, when n is the length of this vector
 */
__global__ void set_to_linspace_cu(int n, int *vec, int i0, int di)
{

  const int num_threads = blockDim.x * gridDim.x;
  const int tid = blockIdx.x * blockDim.x + threadIdx.x;    
  for (int i = tid; i < n; i += num_threads) {
    vec[i] = i0 + i*di;	
  }
}

/** @brief compute cusum from the given pattern*/
__global__ void compute_cusum_cu(int n, int* vec, const double* id)
{
  const int num_threads = blockDim.x * gridDim.x;
  const int tid = blockIdx.x * blockDim.x + threadIdx.x;    
  for (int i = tid; i < n; i += num_threads) {
    if(i==0) {
      vec[i] = 0;
    } else {
      // from i=1..n
      if(id[i-1]!=0.0){
        vec[i] = 1;
      } else {
        vec[i] = 0;        
      }
    }
  }
}

/// @brief Copy the entries in 'dd' where corresponding 'ix' is nonzero, to vd starting at start_index_in_dest.
__global__ void copyToStartingAt_w_pattern_cu(int n_src, 
                                              int n_dest,
                                              int start_index_in_dest,
                                              int* nnz_cumsum, 
                                              double *vd,
                                              const double* dd)
{
  const int num_threads = blockDim.x * gridDim.x;
  const int tid = blockIdx.x * blockDim.x + threadIdx.x;    
  for (int i = tid+1; i < n_src+1; i += num_threads) {
    if(nnz_cumsum[i] != nnz_cumsum[i-1]){
      int idx_dest = nnz_cumsum[i-1] + start_index_in_dest;
      vd[idx_dest] = dd[i-1];
    }
  }
}

namespace hiop
{
namespace cuda
{

constexpr int block_size=256;

/// @brief Copy from src the elements specified by the indices in id. 
void copy_from_index_kernel(int n_local,
                            double* yd,
                            const double* src,
                            const int* id)
{
  int num_blocks = (n_local+block_size-1)/block_size;
  copy_from_index_cu<<<num_blocks,block_size>>>(n_local, yd, src, id);
}

/** @brief Set y[i] = min(y[i],c), for i=[0,n_local-1] */
void component_min_kernel(int n_local,
                          double* yd,
                          double c)
{
  int num_blocks = (n_local+block_size-1)/block_size;
  component_min_cu<<<num_blocks,block_size>>>(n_local, yd, c);
}

/** @brief Set y[i] = min(y[i],x[i], for i=[0,n_local-1] */
void component_min_kernel(int n_local,
                          double* yd,
                          const double* xd)
{
  int num_blocks = (n_local+block_size-1)/block_size;
  component_min_cu<<<num_blocks,block_size>>>(n_local, yd, xd);
}

/** @brief Set y[i] = max(y[i],c), for i=[0,n_local-1] */
void component_max_kernel(int n_local,
                          double* yd,
                          double c)
{
  int num_blocks = (n_local+block_size-1)/block_size;
  component_max_cu<<<num_blocks,block_size>>>(n_local, yd, c);
}

/** @brief Set y[i] = max(y[i],x[i]), for i=[0,n_local-1] */
void component_max_kernel(int n_local,
                          double* yd,
                          const double* xd)
{
  int num_blocks = (n_local+block_size-1)/block_size;
  component_max_cu<<<num_blocks,block_size>>>(n_local, yd, xd);
}

/// @brief Performs axpy, y += alpha*x, on the indexes in this specified by id.
void axpy_w_map_kernel(int n_local,
                       double* yd,
                       const double* xd,
                       const int* id,
                       double alpha)
{
  int num_blocks = (n_local+block_size-1)/block_size;
  axpy_w_map_cu<<<num_blocks,block_size>>>(n_local, yd, xd, id, alpha);
}

/** @brief y[i] += alpha*x[i]*z[i] forall i */
void axzpy_kernel(int n_local,
                  double* yd,
                  const double* xd,
                  const double* zd,
                  double alpha)
{
  int num_blocks = (n_local+block_size-1)/block_size;
  axzpy_cu<<<num_blocks,block_size>>>(n_local, yd, xd, zd, alpha);
}

/** @brief y[i] += alpha*x[i]/z[i] forall i */
void axdzpy_kernel(int n_local,
                   double* yd,
                   const double* xd,
                   const double* zd,
                   double alpha)
{
  int num_blocks = (n_local+block_size-1)/block_size;
  axdzpy_cu<<<num_blocks,block_size>>>(n_local, yd, xd, zd, alpha);
}

/** @brief y[i] += alpha*x[i]/z[i] forall i with pattern selection */
void axdzpy_w_pattern_kernel(int n_local,
                             double* yd,
                             const double* xd,
                             const double* zd,
                             const double* id,
                             double alpha)
{
  int num_blocks = (n_local+block_size-1)/block_size;
  axdzpy_w_pattern_cu<<<num_blocks,block_size>>>(n_local, yd, xd, zd, id, alpha);
}

/** @brief y[i] += c forall i */
void add_constant_kernel(int n_local, double* yd, double c)
{
  int num_blocks = (n_local+block_size-1)/block_size;
  add_constant_cu<<<num_blocks,block_size>>>(n_local, yd, c);
}

/** @brief y[i] += c forall i with pattern selection */
void  add_constant_w_pattern_kernel(int n_local, double* yd, const double* id, double c)
{
  int num_blocks = (n_local+block_size-1)/block_size;
  add_constant_w_pattern_cu<<<num_blocks,block_size>>>(n_local, yd, c, id);
}

/// @brief Invert (1/x) the elements of this
void invert_kernel(int n_local, double* yd)
{
  int num_blocks = (n_local+block_size-1)/block_size;
  invert_cu<<<num_blocks,block_size>>>(n_local, yd);
}

/** @brief y[i] += alpha*1/x[i] + y[i] forall i with pattern selection */
void adxpy_w_pattern_kernel(int n_local,
                            double* yd,
                            const double* xd,
                            const double* id,
                            double alpha)
{
  int num_blocks = (n_local+block_size-1)/block_size;
  adxpy_w_pattern_cu<<<num_blocks,block_size>>>(n_local, yd, xd, id, alpha);
}

/**  @brief  elements of this that corespond to nonzeros in ix are divided by elements of v.
     The rest of elements of this are set to zero.*/
void component_div_w_pattern_kernel(int n_local,
                                    double* yd,
                                    const double* xd,
                                    const double* id)
{
  int num_blocks = (n_local+block_size-1)/block_size;
  component_div_w_pattern_cu<<<num_blocks,block_size>>>(n_local, yd, xd, id);
}

/** @brief Linear damping term */
void set_linear_damping_term_kernel(int n_local,
                                    double* yd,
                                    const double* vd,
                                    const double* ld,
                                    const double* rd)
{
  // compute linear damping term
  int num_blocks = (n_local+block_size-1)/block_size;
  set_linear_damping_term_cu<<<num_blocks,block_size>>>(n_local, yd, vd, ld, rd);
}

/** 
* @brief Performs `this[i] = alpha*this[i] + sign*ct` where sign=1 when EXACTLY one of 
* ixleft[i] and ixright[i] is 1.0 and sign=0 otherwise. 
*/
void add_linear_damping_term_kernel(int n_local,
                                    double* yd,
                                    const double* ixl,
                                    const double* ixr,
                                    double alpha,
                                    double ct)
{
  int num_blocks = (n_local+block_size-1)/block_size;
  add_linear_damping_term_cu<<<num_blocks,block_size>>>(n_local, yd, ixl, ixr, alpha, ct);
}

/** @brief Checks if selected elements of `this` are positive */
void is_posive_w_pattern_kernel(int n_local,
                                double* yd,
                                const double* xd,
                                const double* id)
{
  int num_blocks = (n_local+block_size-1)/block_size;
  is_posive_w_pattern_cu<<<num_blocks,block_size>>>(n_local, yd, xd, id);
}

/// set value with pattern
void set_val_w_pattern_kernel(int n_local,
                              double* yd,
                              const double* xd,
                              const double* id,
                              double max_val)
{
  int num_blocks = (n_local+block_size-1)/block_size;
  set_val_w_pattern_cu<<<num_blocks,block_size>>>(n_local, yd, xd, id, max_val);
}

/** @brief Project solution into bounds  */
void project_into_bounds_kernel(int n_local,
                                double* xd,
                                const double* xld,
                                const double* ild,
                                const double* xud,
                                const double* iud,
                                double kappa1,
                                double kappa2,
                                double small_real)
{
  int num_blocks = (n_local+block_size-1)/block_size;
  project_into_bounds_cu<<<num_blocks,block_size>>>(n_local, xd, xld, ild, xud, iud, kappa1, kappa2, small_real);
}

/** @brief max{a\in(0,1]| x+ad >=(1-tau)x} */
void fraction_to_the_boundry_kernel(int n_local,
                                    double* yd,
                                    const double* xd,
                                    const double* dd,
                                    double tau)
{
  int num_blocks = (n_local+block_size-1)/block_size;
  fraction_to_the_boundry_cu<<<num_blocks,block_size>>>(n_local, yd, xd, dd, tau);
}

/** @brief max{a\in(0,1]| x+ad >=(1-tau)x} with pattern select */
void fraction_to_the_boundry_w_pattern_kernel(int n_local,
                                              double* yd,
                                              const double* xd,
                                              const double* dd,
                                              const double* id,
                                              double tau)
{
  int num_blocks = (n_local+block_size-1)/block_size;
  fraction_to_the_boundry_w_pattern_cu<<<num_blocks,block_size>>>(n_local, yd, xd, dd, id, tau);
}

/** @brief Set elements of `this` to zero based on `select`.*/
void select_pattern_kernel(int n_local, double* yd, const double* id)
{
  int num_blocks = (n_local+block_size-1)/block_size;
  select_pattern_cu<<<num_blocks,block_size>>>(n_local, yd, id);
}

/** @brief y[i] = 0 if id[i]==0.0 && xd[i]!=0.0, otherwise y[i] = 1*/
void component_match_pattern_kernel(int n_local, int* yd, const double* xd, const double* id)
{
  int num_blocks = (n_local+block_size-1)/block_size;
  set_match_pattern_cu<<<num_blocks,block_size>>>(n_local, yd, xd, id);
}

/** @brief Adjusts duals. */
void adjustDuals_plh_kernel(int n_local,
                            double* yd,
                            const double* xd,
                            const double* id,
                            double mu,
                            double kappa)
{
  int num_blocks = (n_local+block_size-1)/block_size;
  adjust_duals_cu<<<num_blocks,block_size>>>(n_local, yd, xd, id, mu, kappa);
}

/// @brief set int array 'arr', starting at `start` and ending at `end`, to the values in `arr_src` from 'start_src`
void set_array_from_to_kernel(int n_local,
                              hiop::hiopInterfaceBase::NonlinearityType* arr, 
                              int start, 
                              int length, 
                              const hiop::hiopInterfaceBase::NonlinearityType* arr_src,
                              int start_src) 
{
  int num_blocks = (n_local+block_size-1)/block_size;
  set_nonlinear_type_cu<<<num_blocks,block_size>>> (n_local, length, arr, start, arr_src, start_src);
}

/// @brief set int array 'arr', starting at `start` and ending at `end`, to the values in `arr_src` from 'start_src`
void set_array_from_to_kernel(int n_local,
                              hiop::hiopInterfaceBase::NonlinearityType* arr, 
                              int start, 
                              int length,
                              hiop::hiopInterfaceBase::NonlinearityType arr_src)
{
  int num_blocks = (n_local+block_size-1)/block_size;
  set_nonlinear_type_cu<<<num_blocks,block_size>>> (n_local, length, arr, start, arr_src);
}

/// @brief Set all elements to c.
void thrust_fill_kernel(int n, double* ptr, double c)
{
  thrust::device_ptr<double> dev_ptr = thrust::device_pointer_cast(ptr);  
  thrust::fill(thrust::device, dev_ptr, dev_ptr+n, c);
}

/** @brief inf norm on single rank */
double infnorm_local_kernel(int n, double* data_dev)
{
  thrust_abs<double> abs_op;
  thrust::maximum<double> max_op;
  thrust::device_ptr<double> dev_ptr = thrust::device_pointer_cast(data_dev);

  // compute one norm
  double norm = thrust::transform_reduce(thrust::device, data_dev, data_dev+n, abs_op, 0.0, max_op);

  return norm;
}

/** @brief Return the one norm */
double onenorm_local_kernel(int n, double* data_dev)
{
  thrust_abs<double> abs_op;
  thrust::plus<double> plus_op;
  thrust::device_ptr<double> dev_ptr = thrust::device_pointer_cast(data_dev);
  //thrust::device_ptr<double> dev_ptr(data_dev);

  // compute one norm
  double norm = thrust::transform_reduce(thrust::device, data_dev, data_dev+n, abs_op, 0.0, plus_op);

  return norm;
}

/** @brief d1[i] = d1[i] * d2[i] forall i */
void thrust_component_mult_kernel(int n, double* d1, const double* d2)
{
  // wrap raw pointer with a device_ptr 
  thrust::multiplies<double> mult_op;
  thrust::device_ptr<double> dev_v1 = thrust::device_pointer_cast(d1);
  thrust::device_ptr<const double> dev_v2 = thrust::device_pointer_cast(d2);
  
  thrust::transform(thrust::device,
                    dev_v1, dev_v1+n,
                    dev_v2, dev_v1,
                    mult_op);
}

/** @brief d1[i] = d1[i] / d2[i] forall i */
void thrust_component_div_kernel(int n, double* d1, const double* d2)
{
  // wrap raw pointer with a device_ptr 
  thrust::divides<double> div_op;
  thrust::device_ptr<double> dev_v1 = thrust::device_pointer_cast(d1);
  thrust::device_ptr<const double> dev_v2 = thrust::device_pointer_cast(d2);
  
  thrust::transform(thrust::device,
                    dev_v1, dev_v1+n,
                    dev_v2, dev_v1,
                    div_op);
}

/** @brief d1[i] = abs(d1[i]) forall i */
void thrust_component_abs_kernel(int n, double* d1)
{
  // wrap raw pointer with a device_ptr 
  thrust_abs<double> abs_op;
  thrust::device_ptr<double> dev_v1 = thrust::device_pointer_cast(d1);
  
  // compute abs
  thrust::transform(thrust::device, dev_v1, dev_v1+n, dev_v1, abs_op);
}

/** @brief d1[i] = sign(d1[i]) forall i */
void thrust_component_sgn_kernel(int n, double* d1)
{
  // wrap raw pointer with a device_ptr 
  thrust_sig<double> sig_op;
  thrust::device_ptr<double> dev_v1 = thrust::device_pointer_cast(d1);
  
  // compute sign
  thrust::transform(thrust::device, dev_v1, dev_v1+n, dev_v1, sig_op);
}

/** @brief d1[i] = sqrt(d1[i]) forall i */
void thrust_component_sqrt_kernel(int n, double* d1)
{
  // wrap raw pointer with a device_ptr 
  thrust_sqrt<double> sqrt_op;
  thrust::device_ptr<double> dev_v1 = thrust::device_pointer_cast(d1);
  
  // compute sqrt
  thrust::transform(thrust::device, dev_v1, dev_v1+n, dev_v1, sqrt_op);
}

/** @brief d1[i] = -(d1[i]) forall i */
void thrust_negate_kernel(int n, double* d1)
{
  thrust::device_ptr<double> dev_v1 = thrust::device_pointer_cast(d1);
  thrust::transform(thrust::device, dev_v1, dev_v1+n, dev_v1, thrust::negate<double>());
}

/** @brief compute sum(log(d1[i])) forall i where id[i]=1*/
double log_barr_obj_kernel(int n, double* d1, const double* id)
{
  thrust::device_ptr<double> dev_v = thrust::device_pointer_cast(d1);
  thrust::device_ptr<const double> id_v = thrust::device_pointer_cast(id);

  // wrap raw pointer with a device_ptr 
  thrust_log_select<double> log_select_op;
  thrust::plus<double> plus_op;
  thrust::multiplies<double> mult_op;
  
  // TODO: how to avoid this temp vec?
  thrust::device_ptr<double> v_temp = thrust::device_malloc(n*sizeof(double));

  // compute x*id
  thrust::transform(thrust::device, dev_v, dev_v+n, id_v, v_temp, mult_op);
  // compute log(y) for y > 0
  double sum = thrust::transform_reduce(thrust::device, v_temp, v_temp+n, log_select_op, 0.0, plus_op);

  thrust::device_free(v_temp);

  return sum;
}

/** @brief compute sum(d1[i]) */
double thrust_sum_kernel(int n, double* d1)
{
  thrust::device_ptr<double> dev_v1 = thrust::device_pointer_cast(d1);
  // compute sum
  return thrust::reduce(thrust::device, dev_v1, dev_v1+n, 0.0, thrust::plus<double>());
}

/** @brief Linear damping term */
double linear_damping_term_kernel(int n,
                                  const double* vd,
                                  const double* ld,
                                  const double* rd,
                                  double mu,
                                  double kappa_d)
{
  // TODO: how to avoid this temp vec?
  thrust::device_vector<double> v_temp(n);
  double* dv_ptr = thrust::raw_pointer_cast(v_temp.data());

  // compute linear damping term
  hiop::cuda::set_linear_damping_term_kernel(n, dv_ptr, vd, ld, rd);

  double term = thrust::reduce(thrust::device, v_temp.begin(), v_temp.end(), 0.0, thrust::plus<double>());

  term *= mu;
  term *= kappa_d;
  return term;
}

/** @brief compute min(d1) */
double min_local_kernel(int n, double* d1)
{
  thrust::device_ptr<double> dev_v1 = thrust::device_pointer_cast(d1);
  thrust::device_ptr<double> ret_dev_ptr = thrust::min_element(thrust::device, dev_v1, dev_v1+n);
  
  double *ret_ptr = thrust::raw_pointer_cast(ret_dev_ptr);
  double *ret_host = new double[1]; 
  hipError_t cuerr = hipMemcpy(ret_host, ret_ptr, (1)*sizeof(double), hipMemcpyDeviceToHost);
 
  double rv = ret_host[0];
  delete [] ret_host;
  
  return rv;
}

/** @brief Checks if selected elements of `this` are positive */
int all_positive_w_pattern_kernel(int n, const double* d1, const double* id)
{
  // TODO: how to avoid this temp vec?
  thrust::device_vector<double> v_temp(n);
  double* dv_ptr = thrust::raw_pointer_cast(v_temp.data());

  hiop::cuda::is_posive_w_pattern_kernel(n, dv_ptr, d1, id);
  
  return thrust::reduce(thrust::device, v_temp.begin(), v_temp.end(), (int)0, thrust::plus<int>());
}

/** @brief compute min(d1) for selected elements*/
double min_w_pattern_kernel(int n, const double* d1, const double* id, double max_val)
{
  // TODO: how to avoid this temp vec?
  thrust::device_ptr<double> dv_ptr = thrust::device_malloc(n*sizeof(double));
  double* d_ptr = thrust::raw_pointer_cast(dv_ptr);

  // set value with pattern
  hiop::cuda::set_val_w_pattern_kernel(n, d_ptr, d1, id, max_val);

  thrust::device_ptr<double> ret_dev_ptr = thrust::min_element(thrust::device, dv_ptr, dv_ptr+n);

  // TODO: how to return double from device to host?
  double *ret_host = new double[1];
  double *ret_ptr = thrust::raw_pointer_cast(ret_dev_ptr);
  hipError_t cuerr = hipMemcpy(ret_host, ret_ptr, (1)*sizeof(double), hipMemcpyDeviceToHost);

  double ret_v = ret_host[0];
  delete [] ret_host;

  thrust::device_free(dv_ptr);
  
  return ret_v;
}

/** @brief check if xld[i] < xud[i] forall i */
bool check_bounds_kernel(int n, const double* xld, const double* xud)
{
  // Perform preliminary check to see of all upper value
  thrust::minus<double> minus_op;
  thrust::device_ptr<double> dev_xud = thrust::device_pointer_cast(const_cast<double*>(xud));
  thrust::device_ptr<double> dev_xld = thrust::device_pointer_cast(const_cast<double*>(xld));

  // TODO: how to avoid this temp vec?
  thrust::device_ptr<double> dv_ptr = thrust::device_malloc(n*sizeof(double));

  thrust::transform(thrust::device,
                    dev_xud, dev_xud+n,
                    dev_xld, dv_ptr,
                    minus_op);  

  int res_offset = thrust::min_element(thrust::device, dv_ptr, dv_ptr + n) - dv_ptr;
  double ret_v = *(dv_ptr + res_offset);
  
  bool bval = (ret_v > 0.0) ? 1 : 0;

  thrust::device_free(dv_ptr);
  
  if(false == bval) 
    return false;

  return true;
}

/** @brief compute max{a\in(0,1]| x+ad >=(1-tau)x} */
double min_frac_to_bds_kernel(int n, const double* xd, const double* dd, double tau)
{
  thrust::device_ptr<double> dv_ptr = thrust::device_malloc(n*sizeof(double));
  double* d_ptr = thrust::raw_pointer_cast(dv_ptr);

  // set values
  hiop::cuda::fraction_to_the_boundry_kernel(n, d_ptr, xd, dd, tau);
  int res_offset = thrust::min_element(thrust::device, dv_ptr, dv_ptr+n) - dv_ptr;
  double alpha = *(dv_ptr + res_offset);

  thrust::device_free(dv_ptr);
  
  return alpha;
}

/** @brief max{a\in(0,1]| x+ad >=(1-tau)x} with pattern id */
double min_frac_to_bds_w_pattern_kernel(int n,
                                        const double* xd,
                                        const double* dd,
                                        const double* id,
                                        double tau)
{
  // TODO: how to avoid this temp vec?
  thrust::device_vector<double> v_temp(n);
  double* dv_ptr = thrust::raw_pointer_cast(v_temp.data());

  // set value with pattern
  hiop::cuda::fraction_to_the_boundry_w_pattern_kernel(n, dv_ptr, xd, dd, id, tau);
  double alpha = *(thrust::min_element(thrust::device, v_temp.begin(), v_temp.end()));

  return alpha;
}

/** @brief Checks if `xd` matches nonzero pattern of `id`. */
bool match_pattern_kernel(int n, const double* xd, const double* id)
{
  // TODO: how to avoid this temp vec?
  thrust::device_vector<int> v_temp(n);
  int* dv_ptr = thrust::raw_pointer_cast(v_temp.data());

  // check if xd matches the pattern given by id
  hiop::cuda::component_match_pattern_kernel(n, dv_ptr, xd, id);

  thrust_istrue istrue_op;

  return thrust::all_of(thrust::device, v_temp.begin(), v_temp.end(), istrue_op);
}

/** @brief Checks if all x[i] = 0 */
bool is_zero_kernel(int n, double* xd)
{
  // wrap raw pointer with a device_ptr 
  thrust_iszero<double> iszero_op;
  thrust::device_ptr<double> dev_v = thrust::device_pointer_cast(xd);

  return thrust::all_of(thrust::device, dev_v, dev_v+n, iszero_op);
}

/** @brief Checks if any x[i] = nan */
bool isnan_kernel(int n, double* xd)
{
  // wrap raw pointer with a device_ptr 
  thrust_isnan<double> isnan_op;
  thrust::device_ptr<double> dev_v = thrust::device_pointer_cast(xd);

  return thrust::any_of(thrust::device, dev_v, dev_v+n, isnan_op);
}

/** @brief Checks if any x[i] = inf */
bool isinf_kernel(int n, double* xd)
{
  // wrap raw pointer with a device_ptr 
  thrust_isinf<double> isinf_op;
  thrust::device_ptr<double> dev_v = thrust::device_pointer_cast(xd);

  return thrust::any_of(thrust::device, dev_v, dev_v+n, isinf_op);
}

/** @brief Checks if all x[i] != inf */
bool isfinite_kernel(int n, double* xd)
{
  // wrap raw pointer with a device_ptr 
  thrust_isfinite<double> isfinite_op;
  thrust::device_ptr<double> dev_v = thrust::device_pointer_cast(xd);

  return thrust::all_of(thrust::device, dev_v, dev_v+n, isfinite_op);
}

/// @brief get number of values that are less than the given value 'val'.
int num_of_elem_less_than_kernel(int n, double* xd, double val)
{
  thrust::device_ptr<double> dev_v = thrust::device_pointer_cast(xd);
  int rval = thrust::transform_reduce(thrust::device, dev_v, dev_v+n, thrust_less(val), (int) 0, thrust::plus<int>());
  return rval;
}

/// @brief get number of values whose absolute value are less than the given value 'val'.
int num_of_elem_absless_than_kernel(int n, double* xd, double val)
{
  thrust::device_ptr<double> dev_v = thrust::device_pointer_cast(xd);
  int rval = thrust::transform_reduce(thrust::device, dev_v, dev_v+n, thrust_abs_less(val), (int) 0, thrust::plus<int>());
  return rval;
}

/// @brief Copy the entries in 'dd' where corresponding 'ix' is nonzero, to vd starting at start_index_in_dest.
void copyToStartingAt_w_pattern_kernel(int n_src, 
                                       int n_dest,
                                       int start_index_in_dest,
                                       int* nnz_cumsum, 
                                       double *vd,
                                       const double* dd)
{
  int num_blocks = (n_src+block_size-1)/block_size;
  copyToStartingAt_w_pattern_cu<<<num_blocks,block_size>>>(n_src,
                                                           n_dest,
                                                           start_index_in_dest,
                                                           nnz_cumsum,
                                                           vd,
                                                           dd);
}



/// for hiopVectorIntCuda
/**
 * @brief Set the vector entries to be a linear space of starting at i0 containing evenly 
 * incremented integers up to i0+(n-1)di, when n is the length of this vector
 */
void set_to_linspace_kernel(int sz, int* buf, int i0, int di)
{
  int num_blocks = (sz+block_size-1)/block_size;
  set_to_linspace_cu<<<num_blocks,block_size>>>(sz, buf, i0, di);
}

/** @brief compute cusum from the given pattern*/
void compute_cusum_kernel(int sz, int* buf, const double* id)
{
  int num_blocks = (sz+block_size-1)/block_size;
  compute_cusum_cu<<<num_blocks,block_size>>>(sz, buf, id);

  thrust::device_ptr<int> dev_v = thrust::device_pointer_cast(buf);
  thrust::inclusive_scan(dev_v, dev_v + sz, dev_v); // in-place scan
}

}

}
